#include "hip/hip_runtime.h"
/* Sim.c
The purpose of this file is to run the game simulations -- Think Main()
Programmer: Nick Kolegraff
Date: 5/29/2011
*/

#include "Player.cu"
#include "cutil_inline.h"
#include <cuPrintf.cu>
#include "stdio.h"
#include "unistd.h" /* hahahah, unistd.... */

// #define GRID_DIMX 52
// #define GRID_DIMY 52
// #define GRID_DIMZ 1
// #define BLOCK_DIMX 52
// #define BLOCK_DIMY 1
// #define BLOCK_DIMZ 1

#define GRID_DIMX 52
#define GRID_DIMY 52
#define GRID_DIMZ 1
#define BLOCK_DIMX 52
#define BLOCK_DIMY 1
#define BLOCK_DIMZ 1

int  *h_results, *d_results;

__device__
void printc(int rank, int suit)
{
  char *ranks = "23456789TJQKA";
  char *suits = "sdch";
  cuPrintf("%c%c ", ranks[rank], suits[suit]);
}

__device__
void printh(CARD *hand, int size)
{
  int i;
  for(i =0; i < size; i++)
    {
      printc(hand[i].rank, hand[i].suit);
    }
  cuPrintf("\n");
}

/* Kernel Method..where the magic happens*/
/* Quora is the shit...just found an answer to a combinatorics function that removes the iteration dependency!!! You sent an email to youself search for combinatorics */
__global__ 
void RunSim(int *results)
{
  PLAYER p;
  InitPlayer(&p);
  int start, i, j, k, l, m, n, rank, size=52;
  int threadsPerBlock = blockDim.x * blockDim.y;
  int threadNumInBlock = threadIdx.x;
  int blockNumInGrid = blockIdx.x + (gridDim.x * blockIdx.y);
  unsigned long long gindex = threadNumInBlock + threadsPerBlock * blockNumInGrid;
  // unsigned long long gdex;
  // for(start = 0; start < 4; start++)
  // if(threadIdx.x < blockIdx.x && threadIdx.x < blockIdx.y && blockIdx.y < blockIdx.x)
  //   {
  for(i = 0; i < size; i++)
    for(j = i+1; j < size; j++)
      for(k = 0; k < size; k++)
  	for(l = k+1; l < size; l++)
  	  // for(m = l+1; m < size; m++)
	  //   for(n = m+1; n < size; n++)

	      {
		if(
		   threadIdx.x != blockIdx.y && threadIdx.x != blockIdx.x && threadIdx.x != i && threadIdx.x != j && threadIdx.x != k && threadIdx.x != l
		   && blockIdx.y != blockIdx.x && blockIdx.y != i && blockIdx.y != j && blockIdx.y != k && blockIdx.y != l
		   && blockIdx.x != i && blockIdx.x != j && blockIdx.x != k && blockIdx.x != l
		   && i != j && i != k && i != l
		   && j != k && j != l
		   && k != l
		   )
		  {

		    p.hand[0].suit = blockIdx.x % 4;
		    p.hand[0].rank = blockIdx.x % 13;
		    // p.hand[0].whos_card = BOARDS;
		    
		    p.hand[1].suit = blockIdx.y % 4;
		    p.hand[1].rank = blockIdx.y % 13;
		    // p.hand[0].whos_card = BOARDS;
		    
		    p.hand[2].suit = threadIdx.x % 4;
		    p.hand[2].rank = threadIdx.x % 13;
		    // p.hand[0].whos_card = BOARDS;
		    
		    p.hand[3].suit = i % 4;
		    p.hand[3].rank = i % 13;
		    // p.hand[0].whos_card = PLAYERS;
		    
		    p.hand[4].suit = j % 4;
		    p.hand[4].rank = j % 13;
		    // p.hand[0].whos_card = PLAYERS;
		    
		    p.hand[5].suit = k % 4;
		    p.hand[5].rank = k % 13;
		    // p.hand[0].whos_card = BOARDS;
		    
		    p.hand[6].suit = l % 4;
		    p.hand[6].rank = l % 13;
		    // p.hand[0].whos_card = BOARDS;

		    rank = rank_hand(p.hand, &p.bin, 7);
		    // atomicAdd(&results[rank], 1);
		  } 
	      }
  // }
}

int main(int argc, char *argv[])
{
  hipEvent_t evt;
  hipEventCreate(&evt);
  cudaPrintfInit();
  int size = 9;

  // initialize memory on host
  printf("Malloc-ing memory on host...");
  h_results = (int *)malloc(size * sizeof(int *));
  int x;
  for(x = 0; x < size; x++)
    {
      h_results[x] = 0;
    }
  printf("Done.\n");

  // copy host memory to device
  printf("Malloc-ing memory on device...");
  hipMalloc(&d_results, size * sizeof(int *));
  hipMemcpy(d_results, h_results, size * sizeof(int *), hipMemcpyHostToDevice);
  printf("Done.\n");
  
  // set up kernel dimensions
  dim3 GRID(GRID_DIMX, GRID_DIMY, GRID_DIMZ);
  dim3 BLOCK(BLOCK_DIMX, BLOCK_DIMY, BLOCK_DIMZ);

  // printf("Executing Kernel.");
  RunSim<<<GRID, BLOCK>>>(d_results);
  cutilCheckMsg("\nKernel execution failed\n");
  
  // Wait for kernel
  hipEventRecord(evt, NULL);
  while(hipEventQuery(evt) == hipErrorNotReady) { usleep(1000000); }

  printf("\nCopying resulsts from device to host...");
  hipMemcpy(h_results, d_results, size * sizeof(int *), hipMemcpyDeviceToHost);
  printf("Done.\n");


  cudaPrintfDisplay(stdout, true);
  hipEventDestroy(evt);  
  int i;
  for(i = 0; i < size; i++)
    {
      printf("Hand Rank %d: %d\n",i,  h_results[i]);
    }
}
